#include "hip/hip_runtime.h"
#include "lab1.h"

//kernel call
__global__ void   mmKernel(TYPE *Md, TYPE *Nd, TYPE *Pd, int mrows, int width, int ncols)
{
   __shared__ TYPE Melem[TILE_WIDTH][TILE_WIDTH];
   __shared__ TYPE Nelem[TILE_WIDTH][TILE_WIDTH];

   //Calculate the row index of the Pd element and M
   int Row = blockIdx.y * blockDim.y + threadIdx.y;
   // Calculate the column idenx of Pd and N
   int Col = blockIdx.x * blockDim.x + threadIdx.x;

   TYPE Pvalue = 0.0;

   int runs = 0;
   if (width % TILE_WIDTH > 0)
   {     
       runs++;
   }

   for (int m = 0; m < (width/TILE_WIDTH) + runs; m++)
   {
      //Row is the row index in P (resultant) and in Md matrices
      if ((Row < mrows) && ((m * TILE_WIDTH + threadIdx.x) < width))
         Melem[threadIdx.y][threadIdx.x] = Md[Row * width + (m * TILE_WIDTH + threadIdx.x)];
      else
      {
         Melem[threadIdx.y][threadIdx.x] = 0.0;
      }

      //Col is the column index in P (resultant) and in Nd matrices
      if ((Col < ncols) && ((m * TILE_WIDTH + threadIdx.y) < width))
         Nelem[threadIdx.y][threadIdx.x] = Nd[Col + ncols * (m * TILE_WIDTH + threadIdx.y)];
      else 
      {
         Nelem[threadIdx.y][threadIdx.x] = 0.0;
      }
      __syncthreads();
 
      for (int k = 0; k < TILE_WIDTH; k++) 
         Pvalue += Melem[threadIdx.y][k] * Nelem[k][threadIdx.x];

      __syncthreads();
   }

   if ((Row < mrows) && (Col < ncols))
      Pd[(Row * ncols) + Col] = Pvalue;
}

matrix_t MMonDevice(matrix_t matrix1, matrix_t matrix2) {

   //number of blocks
   int blocksx, blocksy;

   matrix_t retMatrix;

   TYPE *Md, *Nd, *Pd, *P;
   P = (TYPE *) malloc(matrix1.rows * matrix2.cols * sizeof(TYPE));

   dim3 dimBlock;

   hipMalloc(&Md, matrix1.rows * matrix1.cols * sizeof(TYPE));
   hipMemcpy(Md, matrix1.array, matrix1.rows * matrix1.cols * sizeof(TYPE), hipMemcpyHostToDevice);

   hipMalloc(&Nd, matrix2.rows * matrix2.cols * sizeof(TYPE));
   hipMemcpy(Nd, matrix2.array, matrix2.rows * matrix2.cols * sizeof(TYPE), hipMemcpyHostToDevice);

   hipMalloc(&Pd, matrix1.rows * matrix2.cols * sizeof(TYPE));

   blocksx = matrix2.cols / TILE_WIDTH;
   if (matrix2.cols % TILE_WIDTH > 0)
   {
      blocksx++;
   }
   blocksy = matrix1.rows / TILE_WIDTH;
   if(matrix1.rows % TILE_WIDTH)
   {
      blocksy++;
   }
   printf("Number of x blocks is: %d\n",blocksx); 
   printf("Number of y blocks is: %d\n",blocksy); 

   //invoke kernel
   dim3 dimGrid(blocksx, blocksy);

   //incase resulting matrix is less than 1024
  // if (matrix1.rows * matrix2.cols < TILE_WIDTH * TILE_WIDTH) {
   //   dimBlock.x = matrix2.cols;
    //  dimBlock.y = matrix1.rows;
  // }
  // else {
      dimBlock.x = TILE_WIDTH;
      dimBlock.y = TILE_WIDTH;
  // }

   printf("Launching mrows: %d width: %d ncols: %d\n", matrix1.rows, matrix1.cols, matrix2.cols);

   int runs = 0;
   if (matrix1.cols % TILE_WIDTH > 0)
      runs++;
   printf("Number of tiles will be %d\n", matrix1.cols / TILE_WIDTH + runs);
   mmKernel<<<dimGrid, dimBlock>>>(Md, Nd, Pd, matrix1.rows, matrix1.cols, matrix2.cols);

   //copy back
   hipMemcpy(P, Pd, matrix1.rows * matrix2.cols * sizeof(TYPE), hipMemcpyDeviceToHost);

   hipFree(Md);
   hipFree(Nd);
   hipFree(Pd);

   retMatrix.array = P;
   retMatrix.rows = matrix1.rows;
   retMatrix.cols = matrix2.cols;

   return retMatrix;
}

int main(int argc, char* argv[])
{
   matrix_t matrix1, matrix2, matrix3;

   matrix1 = matrix_read(argv[1]);   
   matrix2 = matrix_read(argv[2]);   

   matrix3 = MMonDevice(matrix1, matrix2);
   print_matrix(matrix3);

   free(matrix1.array);
   free(matrix2.array);
   free(matrix3.array);
   return 0;
}

//reads and parses the matrix
//could we return a pointer to the newly allocated matrix?
//might be faster
matrix_t matrix_read(char* filename)
{

   //stat struct for size of file
   struct stat sb;
   //mmap pointer
   char* file_memory;

   /*matrix we return*/
   matrix_t matrix;

   //file descriptor
   int fd;

   //loop counters
   int j;
   int i;

   //open file for reading
   fd = open (filename, O_RDONLY);
   if(fd==-1)
   {
      fprintf(stderr,"error opening file. exitting\n");
      exit(-1);
   }
   // figure out the size
   if(fstat(fd, &sb)==-1)
   {
      fprintf(stderr,"error fstating file. exitting\n");
      exit(-1);

   }
   //add extra space for the null byte
   file_memory = (char*) mmap(0, sb.st_size+1, PROT_READ | PROT_WRITE, MAP_PRIVATE, fd, 0);
   if(file_memory==MAP_FAILED)
   {

      close (fd);
      fprintf(stderr,"mmap failed. Exitting\n");
      exit(-1);
   }
   //close file descriptor
   close (fd);
   //set null byte
   file_memory[sb.st_size+1]='\0';

   i=0;
   matrix.rows=0;
   matrix.cols=0;

   //count the number of rows and columns in the matrix
   while(file_memory[i]!='\0')
   {
      if ((matrix.rows==0) && file_memory[i]==' ' )
      {
         matrix.cols++;
      }
      if(file_memory[i]=='\n')
      {
         matrix.rows++;
      }
      if(file_memory[i]=='\0')
         break;
      i++;
   }
   //printf("rows: %d ",matrix.rows);
   //printf("columns: %d\n",matrix.cols);

   //malloc for matrix
   matrix.array = (TYPE *)malloc(sizeof(TYPE) * (matrix.rows*matrix.rows));

   if (matrix.array == NULL)
   {
      perror("No space to allocate matrix.");
      exit(1);
   }

   for(i=0;i<matrix.rows;i++)
   {
      for(j=0;j<matrix.rows;j++)
      {
#ifdef DOUBLE
         //copy number into matrix
         matrix.array[i*matrix.rows+j]=strtod(file_memory,&file_memory);
#else 
         matrix.array[i*matrix.rows+j]=strtof(file_memory,&file_memory);
#endif

      }
   }
   //we no longer need mmapped memory
   munmap(file_memory, sb.st_size+1);

   return matrix;

}


//print result matrix to 'result.out'
void print_matrix(matrix_t matrix) {
   int i,j;
   FILE *fp;

   fp = fopen("result.out", "w");

   for (i = 0; i < matrix.rows; i++) {
      for (j = 0; j < matrix.cols; j++) {
#ifdef DOUBLE
         fprintf(fp, "%.2lf ", matrix.array[i * matrix.cols + j]);
#else
         fprintf(fp, "%.2f ", matrix.array[i * matrix.cols + j]);
#endif
      }
      fprintf(fp, "\n");
   }
   fclose(fp);
}

